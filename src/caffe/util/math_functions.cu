#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<int>(const int N, const int alpha, int* X){
// NOT INPLEMENT
}

template <>
void caffe_gpu_scal<unsigned int>(const int N, const unsigned int alpha, unsigned int* X){
// NOT INPLEMENT
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<int>(const int n, const int* x, const int* y, int* out){
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_dot<unsigned>(const int n, const unsigned* x, const unsigned* y, unsigned* out){
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<int>(const int n, const int* x, int* y) {
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_asum<unsigned>(const int n, const unsigned* x, unsigned* y) {
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}
/*
template <typename Dtype>
__global__ void sign_kernel(const int n, const Dtype* x, Dtype* y){
  CUDA_KERNEL_LOOP(index, n){
	y[index] = (Dtype(0) < x[index]) - (x[index] < Dtype(0));
  }
}
*/

template <>
void caffe_gpu_sign<int>(const int N, const int* x, int* y){
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_sign<unsigned int>(const int N, const unsigned int* x, unsigned int* y){
}

template <>
void caffe_gpu_scale<int>(const int n, const int alpha, const int* x, int* y){
// NOT IMPLEMENTED YET
}

template <>
void caffe_gpu_scale<unsigned>(const int n, const unsigned alpha, const unsigned* x, unsigned* y){
// NOT IMPLEMENTED YET
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

// NEED TO BE OPTIMIZED!
__global__ 
void quantize_kernel(const int n, const float left, const float right, const float* X, float* Y){
  CUDA_KERNEL_LOOP(index, n) {
		float x = X[index];
		float eps = 1e-6;
		// float idx = (x<-eps)*floor(log2(-x)) + (x>eps)*floor(log2(x)) + (x>=-eps && x<=eps) * left; 
	    float idx = (x>=0) ? floor(log2(x)) : floor(log2(-x));
        idx = (idx<left) * left + (idx>=left && idx<=right) * idx + (idx>right) * right;
        float sign = (x>=0) ? 1.0f : -1.0f;
        float p_up = sign * x / pow(2.0f, idx) - 1;
        float idx_rand = idx + (p_up >= 0.5);

		Y[index] = (x>=-eps && x<=eps) ? 0 : sign * pow(2.0f, idx_rand);
  }
}

__global__ 
void quantize_kernel(const int n, const double left, const double right, const double* X, double* Y){
  CUDA_KERNEL_LOOP(index, n) {
		double x = X[index];
		double eps = 1e-6;
	    double idx = (x>=0) ? floor(log2(x)) : floor(log2(-x));
        idx = (idx<left) * left + (idx>=left && idx<=right) * idx + (idx>right) * right;
        double sign = (x>=0) ? 1.0 : -1.0;
        double p_up = sign * x / pow(2.0, idx) - 1;
        double idx_rand = idx + (p_up >= 0.5);

		Y[index] = (x>=-eps && x<=eps) ? 0 : sign * pow(2.0, idx_rand);
  }
}

template <>
void caffe_gpu_quantize<int>(const int N, const int left, const int right, const int* X, int* Y) {
// NOT IMPLEMEMTED
}

template <>
void caffe_gpu_quantize<unsigned>(const int N, const unsigned left, const unsigned right, const unsigned* X, unsigned* Y) {
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_quantize<float>(const int N, const float left, const float right, const float* X, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  quantize_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, left, right, X, Y);
}

template <>
void caffe_gpu_quantize<double>(const int N, const double left, const double right, const double* X, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  quantize_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, left, right, X, Y);
}

template <>
void caffe_gpu_acmean<int>(const int N, const int* X, int* ret){
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_acmean<unsigned>(const int N, const unsigned* X, unsigned* ret){
// NOT IMPLEMENTED
}

template <>
void caffe_gpu_acmean<float>(const int N, const float* X, float* ret){

}

template <>
void caffe_gpu_acmean<double>(const int N, const double* X, double* ret){
}


template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void clip_kernel(const int n, Dtype* X) {
  CUDA_KERNEL_LOOP(index, n) {
	const Dtype a = X[index];
    X[index] = (a>1) - (a<-1) + a * (a<=1 && a>=-1) ;
  }
}

template <>
void caffe_gpu_clip<float>(const int N, float* X) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  clip_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, X);
}

template <>
void caffe_gpu_clip<double>(const int N, double* X) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  clip_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, X);
}

template <>
void caffe_gpu_clip<int>(const int N, int* X) {
// NOT IMPLEMENT
}

template <>
void caffe_gpu_clip<unsigned int>(const int N, unsigned int* X) {
// NOT IMPLEMENT
}

template <typename Dtype>
__global__ void ternary_kernel(const int n, const Dtype delta, const Dtype* X, Dtype* Y) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype a = X[index];
    Y[index] = (a > delta) - (a < -delta);
  }
}

template <>
void caffe_gpu_ternary<int>(const int N, const int delta, const int* X, int* Y) {
// NOT IMPLEMENT
}

template <>
void caffe_gpu_ternary<unsigned>(const int N, const unsigned delta, const unsigned* X, unsigned* Y) {
// NOT IMPLEMENT
}

template <>
void caffe_gpu_ternary<float>(const int N, const float delta, const float* X, float* Y) {
// NOT IMPLEMENT
  ternary_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, delta, X, Y);
}

template <>
void caffe_gpu_ternary<double>(const int N, const double delta, const double* X, double* Y) {
// NOT IMPLEMENT
  ternary_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, delta, X, Y);
}

template <typename Dtype>
__global__ void twobit_kernel(const int n, const Dtype* X, Dtype* Y) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype a = X[index];
    Y[index] = -2 * (a < -1) + -1 * (a >= -1 && a <= 0) + 1 * (a > 0 && a <= 1)  + 2 * (a > 1) ;
  }
}

template <>
void caffe_gpu_twobit<int>(const int N, const int* X, int* Y) {
// NOT IMPLEMENT
}

template <>
void caffe_gpu_twobit<unsigned>(const int N, const unsigned* X, unsigned* Y) {
// NOT IMPLEMENT
}

template <>
void caffe_gpu_twobit<float>(const int N, const float* X, float* Y) {
// NOT IMPLEMENT
  twobit_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, X, Y);
}

template <>
void caffe_gpu_twobit<double>(const int N, const double* X, double* Y) {
// NOT IMPLEMENT
  twobit_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(N, X, Y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
